#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// sweep.h
////////////////////////////////////////////////////////////////////////////////
//
// Functions:
//   do_sweep( struct STATE *state )
//
//
////////////////////////////////////////////////////////////////////////////////

#include "sweep-cuda.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "parseargs.h"
#include "common.h"

////////////////////////////////////////////////////////////////////////////////
// functions
////////////////////////////////////////////////////////////////////////////////

__device__ long changes;


__global__
void
do_sweep_cuda(
  const struct FLOATBOX *vbox,
  const struct FLOATBOX *ttbox,
  const struct POINT3D *ttstart,
  const struct FORWARDSTAR * const star,
  const int *numinstar
)
{ 
  int x = blockIdx.x * blockDim.x + threadIdx.x + vbox->imin.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y + vbox->imin.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z + vbox->imin.z;
  
  const struct POINT3D here = p3d( x, y, z );
  
  const float vel_here = boxgetglobal( *vbox, here );
  const float tt_here = boxgetglobal( *ttbox, here );
  
   
  for( int l = 0; l < *numinstar; l++ ) {
    // find point in forward star based on offsets
    const struct POINT3D there = p3daddp3d( here, star[l].pos );
  
    // if 'there' is outside the boundaries, then skip
    if (
      p3disless( there, vbox->omin ) ||
      p3dismore( there, vbox->omax )
    ) {
      continue;
    }
    
    // compute delay from 'here' to 'there' with endpoint average
    const float vel_there = boxgetglobal( *vbox, there );
    const float delay = star[l].halfdistance * (vel_here + vel_there);
  
    // ignore the starting point
    if( p3disnotequal( here, *ttstart ) ) {
      const float tt_there = boxgetglobal( *ttbox, there );
      //printf("%f, %f\n", tt_here, tt_there); 
      
      // if offset point has infinity travel time, then update
      if ((tt_here == INFINITY) && (tt_there == INFINITY)) {
        continue;
      }
  
      if ((tt_here != INFINITY) && (tt_there == INFINITY)) {
        boxputglobal( *ttbox, there, delay + tt_here );
        changes++;
        continue;
      }
  
      if ((tt_here == INFINITY) && (tt_there != INFINITY)) {
        boxputglobal( *ttbox, here, delay + tt_there );
        changes++;
        continue;
      }
  
      if ((tt_here != INFINITY) && (tt_there != INFINITY)) {
        // if a shorter travel time through 'there', update 'here'
        if ((delay + tt_there) < tt_here) {
          boxputglobal( *ttbox, here, delay + tt_there );
  	  changes++;
        }
        // if a shorter travel time through 'here', update 'there'
        else if ((delay + tt_here) < tt_there) {
          boxputglobal( *ttbox, there, delay + tt_here );
  	  changes++;
        }
      }
    }
  }
  
}


void
boxcudacpy_host(
  struct FLOATBOX *box,
  struct FLOATBOX **d_box,
  float **d_flat
)
{
  long flat_size = sizeof(float)*p3dcalcvolume(box->size);
  hipMalloc(d_box, sizeof(struct FLOATBOX));
  hipMalloc(d_flat, flat_size);

  hipMemcpy(*d_box, box, sizeof(struct FLOATBOX), hipMemcpyHostToDevice);
  hipMemcpy(*d_flat, box->flat, flat_size, hipMemcpyHostToDevice);
  hipMemcpy(&((*d_box)->flat), d_flat, sizeof(float*), hipMemcpyHostToDevice);
}

void
boxcudacpy_dev(
  struct FLOATBOX **d_box,
  struct FLOATBOX *box,
  float **d_flat
)
{ 
  long flat_size = sizeof(float) *p3dcalcvolume(box->size);
  hipMemcpy(box->flat, *d_flat, flat_size, hipMemcpyDeviceToHost);
}


extern "C"
long
do_sweep (
  struct STATE *state
)
{ 
  struct FLOATBOX vbox = state->vbox;
  struct FLOATBOX ttbox = state->ttbox;
  struct POINT3D ttstart = state->ttstart;
  const struct FORWARDSTAR * const star = state->star;
  const int numinstar = state->numinstar;

  struct FLOATBOX *d_vbox;
  float *d_vflat;
  boxcudacpy_host(&vbox, &d_vbox, &d_vflat);
  
  struct FLOATBOX *d_ttbox;
  float *d_ttflat;
  boxcudacpy_host(&ttbox, &d_ttbox, &d_ttflat);
   
  struct POINT3D *d_ttstart;
  hipMalloc(&d_ttstart, sizeof(struct POINT3D));
  hipMemcpy(d_ttstart, &ttstart, sizeof(struct POINT3D), hipMemcpyHostToDevice);
  
  struct FORWARDSTAR *d_star;
  hipMalloc(&d_star, sizeof(struct FORWARDSTAR));
  hipMemcpy(d_star, star, sizeof(struct FORWARDSTAR), hipMemcpyHostToDevice);
  
  int *d_numinstar;
  hipMalloc(&d_numinstar, sizeof(int));
  hipMemcpy(d_numinstar, &numinstar, sizeof(int), hipMemcpyHostToDevice);
  
  long anychange = 0;   
  
  hipMemcpyToSymbol(HIP_SYMBOL(changes), &anychange, sizeof(long));

  dim3 grid(61, 61, 3);
  dim3 block(4, 4, 17);

  do_sweep_cuda<<<1, block>>>(d_vbox, d_ttbox, d_ttstart, d_star, d_numinstar);
  
  boxcudacpy_dev(&d_ttbox, &ttbox, &d_ttflat);
  hipMemcpyFromSymbol(&anychange, HIP_SYMBOL(changes), sizeof(long));
 
  printf("CHANGES: %ld\n", anychange); 
 
  hipFree(d_vbox);
  hipFree(d_vflat);
  hipFree(d_ttbox);
  hipFree(d_ttflat);
  hipFree(d_ttstart);
  hipFree(d_star);
  hipFree(d_numinstar);
  
  return anychange;
}

// long
// do_sweep (
//   struct STATE *state
// )
/*int 
main()
{
  // copy some state into local stack memory for fastness
  struct POINT3D omin = {1, -5, 2};
  struct POINT3D omax = {241, 241, 51};
  struct POINT3D imin = {121, 121, 5};
  struct POINT3D imax = {241, 241, 51};

  struct FLOATBOX vbox;
  boxalloc(&vbox, omin, omax, imin, imax);
  vbox.flat[10] = 20.0f;  
  
  struct FLOATBOX ttbox;
  boxalloc(&ttbox, omin, omax, imin, imax);
  ttbox.flat[10] = 5.0f;

  struct FLOATBOX *d_vbox;
  float *d_vflat;
  boxcudacpy_host(&vbox, &d_vbox, &d_vflat);
  
  struct FLOATBOX *d_ttbox;
  float *d_ttflat;
  boxcudacpy_host(&ttbox, &d_ttbox, &d_ttflat);

  printf("0: %f\n", vbox.flat[10]);  
  printf("1: %f\n", ttbox.flat[10]);
  // count how many (if any) values we change
  // long changes = 0;
  // long *d_changes;
  // 
  // hipMalloc(&d_changes, sizeof(long));
  // hipMemcpy(d_changes, &changes, sizeof(long), hipMemcpyHostToDevice);
   
  do_sweep_cuda<<<1, 32>>>(d_vbox, d_ttbox);
  
  boxcudacpy_dev(&d_vbox, &vbox, &d_vflat);
  boxcudacpy_dev(&d_ttbox, &ttbox, &d_ttflat);

  printf("0: %f\n", vbox.flat[10]);
  printf("1: %f\n", ttbox.flat[10]); 
  
  hipFree(d_vbox);
  hipFree(d_vflat);
  boxfree(&vbox);
  hipFree(d_ttbox);
  hipFree(d_ttflat);
  boxfree(&ttbox);
  //hipFree(d_changes); 
 // return changes;
}*/


////////////////////////////////////////////////////////////////////////////////
// END
////////////////////////////////////////////////////////////////////////////////
