#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// sweep.h
////////////////////////////////////////////////////////////////////////////////
//
// Functions:
//   do_sweep( struct STATE *state )
//
//
////////////////////////////////////////////////////////////////////////////////

#include "sweep-cuda.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////////////
// functions
////////////////////////////////////////////////////////////////////////////////

extern "C" long
do_sweep (
  struct STATE *state
)
{
  return 0;
}

/*void
boxcudacpy_host(
  struct FLOATBOX *box,
  struct FLOATBOX **d_box,
  float **d_flat
)
{
  long flat_size = sizeof(float)*p3dcalcvolume(box->size);
  hipMalloc(d_box, sizeof(struct FLOATBOX));
  hipMalloc(d_flat, flat_size);

  hipMemcpy(*d_box, box, sizeof(struct FLOATBOX), hipMemcpyHostToDevice);
  hipMemcpy(*d_flat, box->flat, flat_size, hipMemcpyHostToDevice);
  hipMemcpy(&((*d_box)->flat), d_flat, sizeof(float*), hipMemcpyHostToDevice);
}

void
boxcudacpy_dev(
  struct FLOATBOX **d_box,
  struct FLOATBOX *box,
  float **d_flat
)
{ 
  long flat_size = sizeof(float) *p3dcalcvolume(box->size);
  hipMemcpy(box->flat, *d_flat, flat_size, hipMemcpyDeviceToHost);
}
*/

/*__global__
void
do_sweep_cuda(
  const struct FLOATBOX *vbox,
  const struct FLOATBOX *ttbox
  // const struct POINT3D *ttstart,
  // const struct FORWARDSTAR ** const star,
  // const int *numinstar,
  // long *changes
)
{
 //  *changes = 10; 
  ttbox->flat[10] *= 2; 
  vbox->flat[10] *= 3;

}*/

// long
// do_sweep (
//   struct STATE *state
// )
/*int 
main()
{
  // copy some state into local stack memory for fastness
  struct POINT3D omin = {1, -5, 2};
  struct POINT3D omax = {241, 241, 51};
  struct POINT3D imin = {121, 121, 5};
  struct POINT3D imax = {241, 241, 51};

  struct FLOATBOX vbox;
  boxalloc(&vbox, omin, omax, imin, imax);
  vbox.flat[10] = 20.0f;  
  
  struct FLOATBOX ttbox;
  boxalloc(&ttbox, omin, omax, imin, imax);
  ttbox.flat[10] = 5.0f;

  struct FLOATBOX *d_vbox;
  float *d_vflat;
  boxcudacpy_host(&vbox, &d_vbox, &d_vflat);
  
  struct FLOATBOX *d_ttbox;
  float *d_ttflat;
  boxcudacpy_host(&ttbox, &d_ttbox, &d_ttflat);

  printf("0: %f\n", vbox.flat[10]);  
  printf("1: %f\n", ttbox.flat[10]);
  // count how many (if any) values we change
  // long changes = 0;
  // long *d_changes;
  // 
  // hipMalloc(&d_changes, sizeof(long));
  // hipMemcpy(d_changes, &changes, sizeof(long), hipMemcpyHostToDevice);
   
  do_sweep_cuda<<<1, 32>>>(d_vbox, d_ttbox);
  
  boxcudacpy_dev(&d_vbox, &vbox, &d_vflat);
  boxcudacpy_dev(&d_ttbox, &ttbox, &d_ttflat);

  printf("0: %f\n", vbox.flat[10]);
  printf("1: %f\n", ttbox.flat[10]); 
  
  hipFree(d_vbox);
  hipFree(d_vflat);
  boxfree(&vbox);
  hipFree(d_ttbox);
  hipFree(d_ttflat);
  boxfree(&ttbox);
  //hipFree(d_changes); 
 // return changes;
}*/


////////////////////////////////////////////////////////////////////////////////
// END
////////////////////////////////////////////////////////////////////////////////
