
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void
do_sweep_cuda(
  float *vbox
 // const struct FLOATBOX *ttbox,
 // const struct POINT3D *ttstart,
 // const struct FORWARDSTAR ** const star,
 // const int *numinstar_p,
 // long *changes
)
{
 vbox[threadIdx.x] *= 2;
 /* int numinstar = *numinstar_p;

  const struct POINT3D here = p3d( x, y, z );

  const float vel_here = boxgetglobal( *vbox, here );
  const float tt_here = boxgetglobal( *ttbox, here );  
  

  for( int l = 0; l < numinstar; l++ ) {

    // find point in forward star based on offsets
    const struct POINT3D there = p3daddp3d( here, *star[l].pos );

    // if 'there' is outside the boundaries, then skip
    if (
      p3disless( there, vbox->omin ) ||
      p3dismore( there, vbox->omax )
    ) {
      continue;
    }

    // compute delay from 'here' to 'there' with endpoint average
    const float vel_there = boxgetglobal( *vbox, there );
    const float delay = *star[l].halfdistance * (vel_here + vel_there);

    // ignore the starting point
    if( p3disnotequal( here, ttstart ) ) {

      const float tt_there = boxgetglobal( ttbox, there );

      // if offset point has infinity travel time, then update
      if ((tt_here == INFINITY) && (tt_there == INFINITY)) {
        continue;
      }

      if ((tt_here != INFINITY) && (tt_there == INFINITY)) {
        boxputglobal( *ttbox, there, delay + tt_here );
        *changes++;
        continue;
      }

      if ((tt_here == INFINITY) && (tt_there != INFINITY)) {
        boxputglobal( *ttbox, here, delay + tt_there );
        *changes++;
        continue;
      }

      if ((tt_here != INFINITY) && (tt_there != INFINITY)) {
        // if a shorter travel time through 'there', update 'here'
        if ((delay + tt_there) < tt_here) {
	  boxputglobal( *ttbox, here, delay + tt_there );
	  *changes++;
        }
        // if a shorter travel time through 'here', update 'there'
        else if ((delay + tt_here) < tt_there) {
	  boxputglobal( *ttbox, there, delay + tt_here );
	  *changes++;
        }
      }
    }
  }*/

}

long
do_sweep_cuda_init (
  const int N,
  float *vbox
)
{
  float *d_vbox;
  
  size_t size = N * sizeof(float);

  printf("1,1,1: %f", vbox[0]);
  
  
  hipMalloc(&d_vbox, size); 
  hipMemcpy(d_vbox, vbox, size, hipMemcpyHostToDevice);

  do_sweep_cuda<<<1, 10>>>(d_vbox);
   
  hipMemcpy(vbox, d_vbox, size, hipMemcpyDeviceToHost);
  
  printf("1,1,1: %f", vbox[0]);
  return 0;
}


int
main (
  int argc,
  char *argv[]
)
{
  float vbox[10] = {1,2,3,4,5,6,7,8,9,10};
  do_sweep_cuda_init(10, vbox);  
}
